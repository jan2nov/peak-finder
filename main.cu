
#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void peak_list(int i){	
	for (int j =0; j < 5; j++){
		printf("Kernel launch %d", threadIdx.x);
	}
}

int main(int argc, char* argv[])
{

	hipSetDevice(1);
	peak_list<<<1,1>>>(1);

	printf("\n\nThat's All folks!\n");
	return 0;
}
